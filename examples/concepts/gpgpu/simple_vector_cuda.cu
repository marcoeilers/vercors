#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

/*@
  context get_local_size(0) > 0 && get_local_size(1) == 1 && get_local_size(2) == 1;
  context get_num_groups(0) > 0 && get_num_groups(1) == 1 && get_num_groups(2) == 1;
  context a != NULL && b != NULL && c != NULL;
  context \pointer_length(a) >= size && \pointer_length(b) >= size && \pointer_length(c) >= size;
  context \gtid*2 < size ==> (Perm(&a[\gtid*2], 1\2) ** Perm(&b[\gtid*2], 1\2) ** Perm(&c[\gtid*2], 1\1));
  context \gtid*2+1 < size ==> (Perm(&a[\gtid*2+1], 1\2) ** Perm(&b[\gtid*2+1], 1\2) ** Perm(&c[\gtid*2+1], 1\1));
  ensures \gtid*2 < size ==> c[\gtid*2] == a[\gtid*2] + b[\gtid*2];
  ensures \gtid*2+1 < size ==> c[\gtid*2+1] == a[\gtid*2+1] + b[\gtid*2+1];
@*/
__global__ void addArrays(int* a, int* b, int* c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (2*tid+1 < size) {
        int2 a2 = make_int2(a[2*tid], a[2*tid+1]);
        int2 b2 = make_int2(b[2*tid], b[2*tid+1]);
        int2 c2 = make_int2(a2.x+b2.x, a2.y+b2.y);
        c[2*tid] = c2.x;
        c[2*tid+1] = c2.y;
    } else if(2*tid+1 == size) {
        c[2*tid] = a[2*tid] + b[2*tid];
    }
}

